
#include <hip/hip_runtime.h>
#include <thread>
#include <assert.h>
#include <chrono>
#include <iostream>
using namespace std::chrono;
using namespace std;

#define SIZE (1024*1024)
#define OUTER (128)

__global__ void vector_add(float * d_a, float * d_b, float * d_c, int size) {
  int chunk = size / blockDim.x;
  int start = chunk * threadIdx.x;
  int end = start + chunk;
  for (int j = 0; j < OUTER; j++) {
    for (int i = start; i < end; i++) {
      d_a[i] = d_b[i] + d_c[i];
    }
  }
}


int main() {

  float *a = new float[SIZE];
  float *b = new float[SIZE];
  float *c = new float[SIZE];

  for(int i = 0; i < SIZE; i++) {
    a[i] = 0;
    b[i] = i;
    c[i] = 1;
  }

  float *d_a, *d_b, *d_c;
  int e = 0;
  e = hipMalloc(&d_a, SIZE*sizeof(float));
  e |= hipMalloc(&d_b, SIZE*sizeof(float));
  e |= hipMalloc(&d_c, SIZE*sizeof(float));

  e |= hipMemcpy(d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_c, c, SIZE*sizeof(float), hipMemcpyHostToDevice);

  
  auto time_start = high_resolution_clock::now();
  vector_add<<<1,1024>>>(d_a, d_b, d_c, SIZE);
  e |= hipDeviceSynchronize();

  auto time_end = high_resolution_clock::now();
  auto time_duration = duration_cast<nanoseconds>(time_end - time_start);
  double time_seconds = time_duration.count()/1000000000.0;
  
  cout << "timings: " << time_seconds << endl;

  e |= hipMemcpy(a, d_a, SIZE*sizeof(float), hipMemcpyDeviceToHost);

  assert(e == 0);
      
  for (int i = 0; i < SIZE; i++) {
    assert(a[i] == (float(i) + 1.0f));    
  }  

  return 0;
}


  
